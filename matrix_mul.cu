#include "hip/hip_runtime.h"
#include"matmul_utils.hpp"

#define M 3
#define N 3
#define P 3
#define Q 3

__global__ void matmulKernel(int* a, int*b, int* c)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    c[i*Q + j] = 0;

    for(int k=0; k<N; ++k)
    {
        c[i*Q + j] += (a[i*N + k] * b[k*Q + j]);
    }
}

int main()
{
    int* a = new int[M * N * sizeof(int)];
    int* b = new int[P * Q * sizeof(int)];
    int* c = new int[M * Q * sizeof(int)];
    init_matrix(a, M, N);
    init_matrix(b, P, Q);

    std::cout<<"A =\n";
    display_matrix(a, M, N);
    std::cout<<"B =\n";
    display_matrix(b, P, Q);

    assert(N==P); 

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, M * N * sizeof(int));
    hipMalloc((void**)&d_b, P * Q * sizeof(int));
    hipMalloc((void**)&d_c, M * Q * sizeof(int));

    hipMemcpy(d_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, P * Q * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid(M, Q);

    matmulKernel<<<grid, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, M * Q * sizeof(int), hipMemcpyDeviceToHost);
    std::cout<<"A*B =\n";
    display_matrix(c, M, Q);

    delete a;
    delete b;
    delete c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
